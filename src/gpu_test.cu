#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "eventReader.h"
#include "patternReader.h"
#include "tools.h"

#include "gpu_test.h"

using namespace std;

void copyContextToGpu(const PatternContainer& p, const EventContainer& e, GpuContext& ctx);
void runTestKernel(const PatternContainer& p, const EventContainer& e, GpuContext& ctx);
void deleteGpuContext(GpuContext& ctx);
__global__ void testKernel(const int *hashId, const unsigned int *hashIdEventBegin, int *hashId_out, int N);


void copyContextToGpu(const PatternContainer& p, const EventContainer& e, GpuContext& ctx) {
    hipError_t err = hipSuccess;

    // For all group/event begins, calculate indices that are pointed to
    vector<unsigned int> h_hitArrayGroupIndices = pointerToIndex(p.hitArrayGroupBegin, p.hitArray);
    vector<unsigned int> h_hashIdEventIndices = pointerToIndex(e.hashIdEventBegin, e.hashId);
    vector<unsigned int> h_nHitsEventIndices = pointerToIndex(e.nHitsEventBegin, e.nHits);
    vector<unsigned int> h_hitDataEventIndices = pointerToIndex(e.hitDataEventBegin, e.hitData);

    // Calculate size for all arrays that will be transferred
    size_t hashId_array_size = sizeof(int)*p.hashId_array.size();
    size_t hitArray_size = sizeof(unsigned char)*p.hitArray.size();
    size_t hitArrayGroupIndices_size = sizeof(unsigned int)*h_hitArrayGroupIndices.size();
    size_t hashId_size = sizeof(int)*e.hashId.size();
    size_t hashIdEventIndices_size = sizeof(unsigned int)*h_hashIdEventIndices.size();
    size_t nHits_size = sizeof(unsigned int)*e.nHits.size();
    size_t nHitsEventIndices_size = sizeof(unsigned int)*h_nHitsEventIndices.size();
    size_t hitData_size = sizeof(unsigned char)*e.hitData.size();
    size_t hitDataEventIndices_size = sizeof(unsigned int)*h_hitDataEventIndices.size();
    size_t nEventMatches_size = sizeof(int)*e.header.nEvents;

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Allocate space for arrays on device
    err = hipMalloc((void ** )&ctx.d_hashId_array, hashId_array_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hashId_array\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitArray, hitArray_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitArray\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitArrayGroupIndices, hitArrayGroupIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitArrayGroupIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hashId, hashId_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hashId\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hashIdEventIndices, hashIdEventIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hashIdEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_nHits, nHits_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_nHits\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_nHitsEventIndices, nHitsEventIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_nHitsEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitData, hitData_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitData\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitDataEventIndices, hitDataEventIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitDataEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_nEventMatches, nEventMatches_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_nEventMatches\n" << hipGetErrorString(err) << endl;

    // Copy input arrays to device
    err = hipMemcpy(ctx.d_hashId_array, &p.hashId_array[0], hashId_array_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hashId_array not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitArray, &p.hitArray[0], hitArray_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitArray not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitArrayGroupIndices, &h_hitArrayGroupIndices[0], hitArrayGroupIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitArrayGroupIndices not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hashId, &e.hashId[0], hashId_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hashId not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hashIdEventIndices, &h_hashIdEventIndices[0], hashIdEventIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hashIdEventIndices not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_nHits, &e.nHits[0], nHits_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: nHits not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_nHitsEventIndices, &h_nHitsEventIndices[0], nHitsEventIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: nHitsEventIndices not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitData, &e.hitData[0], hitData_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitData not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitDataEventIndices, &h_hitDataEventIndices[0], hitDataEventIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitDataEventIndices not copied to device\n" << hipGetErrorString(err) << endl;

    // Initialise output arrays
    err = hipMemset(ctx.d_nEventMatches, 0, nEventMatches_size);
    if (err != hipSuccess) cerr << "Error: d_nEventMatches not initialised to zero" << endl;

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;
    cout << "Allocated and copied arrays to device in " << msecTotal << " ms" << endl;
};


void runTestKernel(const PatternContainer& p, const EventContainer& e, GpuContext& ctx) {
    hipError_t err = hipSuccess;

    // Calculate number of threads/blocks required
    int N = e.header.nEvents;
    int threadsPerBlock = 256;
    int blocksPerGrid = (N/threadsPerBlock) + 1; 

    // Allocate and initialise vector to store result
    size_t nEventMatches_size = sizeof(int)*e.header.nEvents;
    vector<int> nEventMatches(nEventMatches_size);

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Run the kernel nRepeats times
    int nRepeats = 100;
    for (int i = 0; i < nRepeats; i++) {
        testKernel<<<blocksPerGrid, threadsPerBlock>>>(ctx.d_hashId, ctx.d_hashIdEventIndices, ctx.d_nEventMatches, N);
    }

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;
    cout << "Ran kernel " << nRepeats << " times in " << msecTotal << " ms" << endl;
    float msecPerEvent = msecTotal/nRepeats;
    cout << "Average kernel time is " << msecPerEvent << " ms" << endl;

    // Copy result back to host memory
    err = hipMemcpy(&nEventMatches[0], ctx.d_nEventMatches, nEventMatches_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) cerr << "Error: d_nEventMatches not copied from device to host" << endl;

};


void deleteGpuContext(GpuContext& ctx) {
    hipError_t err = hipSuccess;

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Free device memory
    err = hipFree(ctx.d_hashId_array);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hashId_array\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitArray);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitArray\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitArrayGroupIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitArrayGroupIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hashId);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hashId\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hashIdEventIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hashIdEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_nHits);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_nHits\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_nHitsEventIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_nHitsEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitData);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitData\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitDataEventIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitDataEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_nEventMatches);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_nEventMatches\n" << hipGetErrorString(err) << endl;

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;

    cout << "Freed device memory in " << msecTotal << " ms" << endl;

};


__global__ void testKernel(const int *hashId, const unsigned int *hashIdEventBegin, int *nEventMatches, int N) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N) {
        int nColl = hashIdEventBegin[i+1] - hashIdEventBegin[i];
        for (int j = 0; j < nColl; j++) {
              if (hashId[hashIdEventBegin[i] + j] == 3005) {
                  nEventMatches[i]++;
              }
        }
    }

}
