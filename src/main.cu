#include <string>
#include <iostream>
#include <getopt.h>
#include <hip/hip_runtime.h>

#include "eventReader.h"
#include "patternReader.h"
#include "matchPatterns.h"
#include "gpu_test.h"

using namespace std;

int main(int argc, char* argv[]) {


    string patternFile = "inputs/pattern_groups.bin";
    string eventFile = "inputs/single_track_hit_events.bin";
    char opt;

    while((opt = getopt(argc,argv,"e:p:")) != -1) {
        switch(opt)
        {
            case 'e': eventFile = optarg; break;
            case 'p': patternFile = optarg; break;
            default: cerr << "Invalid argument" << endl; exit(EXIT_FAILURE);
        }
    }

    PatternContainer p;
    readPatterns(patternFile, p);
    //printPatterns(p);

    EventContainer e;
    readEvents(eventFile, e);
    //printEvents(e);

    //matchByEvents(p, e);
    //matchByPatterns(p, e);

    GpuContext ctx;
    copyContextToGpu(p, e, ctx);
    runTestKernel(p, e, ctx);
    deleteGpuContext(ctx);

    return 0;
}
