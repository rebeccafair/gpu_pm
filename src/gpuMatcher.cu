#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "eventReader.h"
#include "patternReader.h"
#include "matchResults.h"
#include "tools.h"

#include "gpuMatcher.h"

using namespace std;

void createGpuContext(const PatternContainer& p, const EventContainer& e, GpuContext& ctx) {
    hipError_t err = hipSuccess;

    // For all group/event begins, calculate indices that are pointed to
    vector<unsigned int> h_hitArrayGroupIndices = pointerToIndex(p.hitArrayGroupBegin, p.hitArray);
    vector<unsigned int> h_hashIdEventIndices = pointerToIndex(e.hashIdEventBegin, e.hashId);
    vector<unsigned int> h_nHitsEventIndices = pointerToIndex(e.nHitsEventBegin, e.nHits);
    vector<unsigned int> h_hitDataEventIndices = pointerToIndex(e.hitDataEventBegin, e.hitData);

    // Calculate size for all arrays that will be transferred
    size_t hashId_array_size = sizeof(int)*p.hashId_array.size();
    size_t hitArray_size = sizeof(unsigned char)*p.hitArray.size();
    size_t hitArrayGroupIndices_size = sizeof(unsigned int)*h_hitArrayGroupIndices.size();
    size_t hashId_size = sizeof(int)*e.hashId.size();
    size_t hashIdEventIndices_size = sizeof(unsigned int)*h_hashIdEventIndices.size();
    size_t nHits_size = sizeof(unsigned int)*e.nHits.size();
    size_t nHitsEventIndices_size = sizeof(unsigned int)*h_nHitsEventIndices.size();
    size_t hitData_size = sizeof(unsigned char)*e.hitData.size();
    size_t hitDataEventIndices_size = sizeof(unsigned int)*h_hitDataEventIndices.size();
    size_t matchingPattIds_size = sizeof(int)*10000;

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Allocate space for arrays on device
    err = hipMalloc((void ** )&ctx.d_hashId_array, hashId_array_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hashId_array\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitArray, hitArray_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitArray\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitArrayGroupIndices, hitArrayGroupIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitArrayGroupIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hashId, hashId_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hashId\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hashIdEventIndices, hashIdEventIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hashIdEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_nHits, nHits_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_nHits\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_nHitsEventIndices, nHitsEventIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_nHitsEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitData, hitData_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitData\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitDataEventIndices, hitDataEventIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitDataEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_matchingPattIds, matchingPattIds_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_matchingPattIds\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_nMatches, sizeof(int));
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_nMatches\n" << hipGetErrorString(err) << endl;

    // Copy input arrays to device
    err = hipMemcpy(ctx.d_hashId_array, &p.hashId_array[0], hashId_array_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hashId_array not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitArray, &p.hitArray[0], hitArray_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitArray not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitArrayGroupIndices, &h_hitArrayGroupIndices[0], hitArrayGroupIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitArrayGroupIndices not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hashId, &e.hashId[0], hashId_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hashId not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hashIdEventIndices, &h_hashIdEventIndices[0], hashIdEventIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hashIdEventIndices not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_nHits, &e.nHits[0], nHits_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: nHits not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_nHitsEventIndices, &h_nHitsEventIndices[0], nHitsEventIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: nHitsEventIndices not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitData, &e.hitData[0], hitData_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitData not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitDataEventIndices, &h_hitDataEventIndices[0], hitDataEventIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitDataEventIndices not copied to device\n" << hipGetErrorString(err) << endl;

    // Initialise output arrays
    err = hipMemset(ctx.d_matchingPattIds, 0, matchingPattIds_size);
    if (err != hipSuccess) cerr << "Error: d_matchingPattIds not initialised to zero" << endl;
    err = hipMemset(ctx.d_nMatches, 0, sizeof(int));
    if (err != hipSuccess) cerr << "Error: d_matchingPattIds not initialised to zero" << endl;

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;
    cout << "Allocated and copied arrays to device in " << msecTotal << " ms" << endl;
};


void runMatchByBlockSingle(const PatternContainer& p, const EventContainer& e, GpuContext& ctx, MatchResults& mr, int threadsPerBlock) {
    hipError_t err = hipSuccess;

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Calculate number of blocks required
    int blocksPerGrid = p.header.nGroups;

    // Run kernel for each event
    int nPattMatchesSize = threadsPerBlock/p.header.nLayers*sizeof(unsigned int);
    for (int i = 0; i < e.header.nEvents; i++ ) {
    //for (int i = 100; i < 125; i++ ) {
        matchByBlockSingle<<<blocksPerGrid, threadsPerBlock, nPattMatchesSize>>>(ctx.d_hashId_array, ctx.d_hitArray, ctx.d_hitArrayGroupIndices,
                                                                                 ctx.d_hashId, ctx.d_hashIdEventIndices, ctx.d_nHits,
                                                                                 ctx.d_nHitsEventIndices, ctx.d_hitData, ctx.d_hitDataEventIndices,
                                                                                 ctx.d_matchingPattIds, ctx.d_nMatches, i);
    }
    hipDeviceSynchronize();

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;
    cout << "Ran kernel " << e.header.nEvents << " times in " << msecTotal << " ms" << endl;
    float msecPerEvent = msecTotal/e.header.nEvents;
    cout << "Average matchByBlockSingle kernel time with " << threadsPerBlock << " threads is " << msecPerEvent << " ms" << endl;

    // Copy result back to host memory
    err = hipMemcpy(&mr.nMatches, ctx.d_nMatches, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) cerr << "Error: d_nMatches not copied from device to host" << endl;
    mr.patternIds.resize(mr.nMatches);
    err = hipMemcpy(&mr.patternIds[0], ctx.d_matchingPattIds, mr.nMatches*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) cerr << "Error: d_matchingPattIds not copied from device to host" << endl;
};

void runMatchByBlockMulti(const PatternContainer& p, const EventContainer& e, GpuContext& ctx, MatchResults& mr, int threadsPerBlock, int nBlocks) {
    hipError_t err = hipSuccess;

    // Distribute groups to blocks according to number of blocks
    vector<int> blockBegin(nBlocks,-1);
    vector<int> nGroupsInBlock(nBlocks,0);
    vector<int> groups(p.header.nGroups,-1);
    distributeWork(nBlocks, p, blockBegin, nGroupsInBlock, groups);

    // Allocate and copy information about block/group assignments to device
    size_t blockBegin_size = sizeof(int)*blockBegin.size();
    err = hipMalloc((void ** )&ctx.d_blockBegin, blockBegin_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_blockBegin\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_blockBegin, &blockBegin[0], blockBegin_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: blockBegin not copied to device\n" << hipGetErrorString(err) << endl;
    size_t nGroupsInBlock_size = sizeof(int)*nGroupsInBlock.size();
    err = hipMalloc((void ** )&ctx.d_nGroupsInBlock, nGroupsInBlock_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_nGroupsInBlock\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_nGroupsInBlock, &nGroupsInBlock[0], nGroupsInBlock_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: nGroupsInBlock not copied to device\n" << hipGetErrorString(err) << endl;
    size_t groups_size = sizeof(int)*groups.size();
    err = hipMalloc((void ** )&ctx.d_groups, groups_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_groups\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_groups, &groups[0], groups_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: groups not copied to device\n" << hipGetErrorString(err) << endl;

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Run kernel for each event
    int nPattMatchesSize = threadsPerBlock/p.header.nLayers*sizeof(unsigned int);
    for (int i = 0; i < e.header.nEvents; i++ ) {
        matchByBlockMulti<<<nBlocks, threadsPerBlock, nPattMatchesSize>>>(ctx.d_hashId_array, ctx.d_hitArray, ctx.d_hitArrayGroupIndices,
                                                                          ctx.d_hashId, ctx.d_hashIdEventIndices, ctx.d_nHits,
                                                                          ctx.d_nHitsEventIndices, ctx.d_hitData, ctx.d_hitDataEventIndices,
                                                                          ctx.d_matchingPattIds, ctx.d_nMatches, i, ctx.d_blockBegin,
                                                                          ctx.d_nGroupsInBlock, ctx.d_groups);
    }
    hipDeviceSynchronize();

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;
    cout << "Ran kernel " << e.header.nEvents << " times in " << msecTotal << " ms" << endl;
    float msecPerEvent = msecTotal/e.header.nEvents;
    cout << "Average matchByBlockMulti kernel time with " << threadsPerBlock << " threads and " << nBlocks << " blocks is " << msecPerEvent << " ms" << endl;

    // Copy result back to host memory
    err = hipMemcpy(&mr.nMatches, ctx.d_nMatches, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) cerr << "Error: d_nMatches not copied from device to host" << endl;
    mr.patternIds.resize(mr.nMatches);
    err = hipMemcpy(&mr.patternIds[0], ctx.d_matchingPattIds, mr.nMatches*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) cerr << "Error: d_matchingPattIds not copied from device to host" << endl;

};

void distributeWork(int nBlocks, const PatternContainer& p, vector<int>& blockBegin, vector<int>& nGroupsInBlock, vector<int>& groups) {

    int maxPattsInBlock = (p.header.nHitPatt + nBlocks - 1)/nBlocks;

    cout << "Distributing work..." << endl;

    vector<int> nPattsInBlock(nBlocks,0);
    vector<int> assignedBlock(p.header.nGroups,-1);

    bool forward = true;
    int nextBlock = 0;
    // Loop through groups and determine number of groups in each block
    for (int g = 0; g < p.header.nGroups; g++) {
       // Loop over blocks and add group to block if number of patterns
       // in block is less than threshold. Alternately loop forward and
       // backward across patterns for better load balancing
       while ( assignedBlock[g] == -1) {
           if (forward) {
               for (int b = nextBlock; b < nBlocks; b++) {
                   if (nPattsInBlock[b] < maxPattsInBlock) {
                       nGroupsInBlock[b]++;
                       nPattsInBlock[b] += p.nPattInGrp[g];
                       assignedBlock[g] = b;
                       if (b != nBlocks - 1) { nextBlock = b + 1; } else { forward = !forward; }
                       break;
                   }
                   if (b == nBlocks - 1) { forward = !forward; } // Reverse loop if loop is completed with no assignments
               }
           } else {
               for (int b = nextBlock; b >= 0; b--) {
                   if (nPattsInBlock[b] < maxPattsInBlock) {
                       nGroupsInBlock[b]++;
                       nPattsInBlock[b] += p.nPattInGrp[g];
                       assignedBlock[g] = b;
                       if (b != 0) { nextBlock = b - 1; } else { forward = !forward; }
                       break;
                   }
                   if (b == 0) { forward = !forward; } // Reverse loop if loop is completed with no assignments
               }
           }
        }

    } // End loop over groups


    // Loop through blocks and point to first index in each group
    int nextIndex = 0;
    for (int b = 0; b < nBlocks; b++) {
        blockBegin[b] = nextIndex;
        nextIndex += nGroupsInBlock[b];
        nGroupsInBlock[b] = 0;
    }

    // Loop through groups again and assign them to correct block
    forward = true;
    nextBlock = 0;
    for (int g = 0; g < p.header.nGroups; g++) {
        int block = assignedBlock[g];
        groups[blockBegin[block] + nGroupsInBlock[block]] = g;
        nGroupsInBlock[block]++;
    }

    // Print assigned blocks
    /*for (int b = 0; b < nBlocks; b++) {
        cout << "block: " << b << " nGroupsInBlock: " << nGroupsInBlock[b] << " nPattsInBlock: " << nPattsInBlock[b] << endl;
        cout << "groups: ";
        for (int g = 0; g < nGroupsInBlock[b]; g++) {
            cout << groups[blockBegin[b] + g] << " ";
        }
        cout << endl;
    }
    */

};

void runMatchByLayer(const PatternContainer& p, const EventContainer& e, GpuContext& ctx, MatchResults& mr, int threadsPerBlock) {
    hipError_t err = hipSuccess;

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Calculate number of blocks required
    int blocksPerGrid = p.header.nGroups;

    // Run kernel for each event
    int nPattMatchesSize = p.nPattInGrp[0]*sizeof(unsigned int);
    for (int i = 0; i < e.header.nEvents; i++ ) {
        matchByLayer<<<blocksPerGrid, threadsPerBlock, nPattMatchesSize>>>(ctx.d_hashId_array, ctx.d_hitArray, ctx.d_hitArrayGroupIndices,
                                                                           ctx.d_hashId, ctx.d_hashIdEventIndices, ctx.d_nHits,
                                                                           ctx.d_nHitsEventIndices, ctx.d_hitData, ctx.d_hitDataEventIndices,
                                                                           ctx.d_matchingPattIds, ctx.d_nMatches, i);
    }
    hipDeviceSynchronize();

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;
    cout << "Ran kernel " << e.header.nEvents << " times in " << msecTotal << " ms" << endl;
    float msecPerEvent = msecTotal/e.header.nEvents;
    cout << "Average matchByLayer kernel time with " << threadsPerBlock << " threads is " << msecPerEvent << " ms" << endl;

    // Copy result back to host memory
    err = hipMemcpy(&mr.nMatches, ctx.d_nMatches, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) cerr << "Error: d_nMatches not copied from device to host" << endl;
    mr.patternIds.resize(mr.nMatches);
    err = hipMemcpy(&mr.patternIds[0], ctx.d_matchingPattIds, mr.nMatches*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) cerr << "Error: d_matchingPattIds not copied from device to host" << endl;

};


void deleteGpuContext(GpuContext& ctx) {
    hipError_t err = hipSuccess;

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Free device memory
    err = hipFree(ctx.d_hashId_array);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hashId_array\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitArray);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitArray\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitArrayGroupIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitArrayGroupIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hashId);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hashId\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hashIdEventIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hashIdEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_nHits);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_nHits\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_nHitsEventIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_nHitsEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitData);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitData\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitDataEventIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitDataEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_matchingPattIds);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_matchingPattIds\n" << hipGetErrorString(err) << endl;

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;

    cout << "Freed device memory in " << msecTotal << " ms" << endl;

};

__global__ void matchByBlockSingle(const int *hashId_array, const unsigned char *hitArray, const unsigned int *hitArrayGroupIndices, 
                                   const int *hashId, const unsigned int *hashIdEventIndices, const unsigned int *nHits,
                                   const unsigned int *nHitsEventIndices, const unsigned char *hitData, const unsigned int *hitDataEventIndices, 
                                   int *matchingPattIds, int *nMatches, const int eventId) {
    const int nLayers = 8;
    const int nRequiredMatches = 7;
    const int nMaxRows = 11;
    const int nMaxColumns = 3;
    const int maxDcBits = 2;
    int grp = blockIdx.x;
    int lyr = threadIdx.x%nLayers;
    int row = threadIdx.x/nLayers;
    int lyrHashId = hashId_array[grp*nLayers + lyr];

    __shared__ unsigned int nHashMatches; // Number of group hashIds that match event collection hashIds
    __shared__ int matchingCollections[nLayers]; // Records which collection matches the group hashId of a certain layer
    __shared__ unsigned int collectionHit[(maxDcBits + 1)*nLayers]; // Records hits as a bit array for a matching collection

    if (threadIdx.x == 0) {
        nHashMatches = 0;
    }
    __syncthreads();

    // Get first nLayers threads to check if any hashIds are wildcards
    if (threadIdx.x < nLayers) {
        // Initialise matchingCollections to -1
        matchingCollections[lyr] = -1;
        // Automatically match if layer is wildcard
        if (lyrHashId == -1) {
            atomicAdd(&nHashMatches,1);
         }
    }

    // Get each thread to compare one hashId with one collection from
    // an event to check for a match
    int nColl = hashIdEventIndices[eventId+1] - hashIdEventIndices[eventId];
    int nLoops = ((nLayers*nColl)/blockDim.x) + 1;
    for (int n = 0; n < nLoops; n++) {
        int coll = n*blockDim.x/nLayers + threadIdx.x/nLayers;
        if (coll < nColl) {
            if (lyrHashId != -1) {
                if (hashId[hashIdEventIndices[eventId] + coll] == lyrHashId) {
                    atomicExch(&matchingCollections[lyr],coll);
                    atomicAdd(&nHashMatches,1);
                }
            }
        }
    }
    __syncthreads();

    if (nHashMatches >= nRequiredMatches) {

        // Initialise collectionHit[]
        if (threadIdx.x < nLayers*(maxDcBits + 1)) {
            collectionHit[threadIdx.x] = 0;
        }
        __syncthreads();

        // Loop through collection hits to find collection hit data
        int matchingColl = matchingCollections[lyr];
        const unsigned char *pHitData = &hitData[hitDataEventIndices[eventId]];
        for (int coll = 0; coll < matchingColl; coll++) {
            pHitData += nHits[nHitsEventIndices[eventId] + coll];
        }

        // Put hits into bit arrays
        unsigned char isPixel = ((*pHitData >> 7) & 1); // If bit 7 is 1, element is pixel, otherwise strip
        if (threadIdx.x < nLayers) {
            if (matchingColl != -1) {
                for (int hit = 0; hit < nHits[nHitsEventIndices[eventId] + matchingColl]; hit++) {
                    unsigned char hitPos = ((*(pHitData + hit) >> 2) & 31); // Get superstrip value if strip, or row value if pixel. Occupies bits 2-6
                    for (int dcBits = 0; dcBits <= maxDcBits; dcBits++) {
                        unsigned char maskedHitPos = hitPos &  (~((1 << dcBits) - 1)); // Mask hit value depending on dc bits. e.g. if dcBits = 2 AND with 11100
                        if (isPixel) { 
                            maskedHitPos += (*(pHitData + hit) & 3)*nMaxRows; // If pixel, need to adjust value according to column
                        }
                        unsigned char bitMask = 1; // Get bitmask  e.g. ...01111 for dcBits = 2
                        if (dcBits == 1) {
                            bitMask = 3;
                        } else if (dcBits == 2) {
                            bitMask = 15;
                        }
                        collectionHit[dcBits*nLayers + lyr] |= (bitMask << maskedHitPos); 
                    }
                }
            }
        }

        // Loop as many times as necessary for all threads to cover all patterns
        int nPattInGrp = (hitArrayGroupIndices[grp + 1] - hitArrayGroupIndices[grp])/nLayers;
        nLoops = ((nPattInGrp*nLayers)/blockDim.x) + 1;
        extern __shared__ unsigned int nPattMatches[];

        for (int n = 0; n < nLoops; n++) {

            int pattNum = n*blockDim.x/nLayers + row;

            // Initialise nPattMatches to zero
            if (lyr == 0) {
                nPattMatches[row] = 0;
            }
            __syncthreads();

            // Only continue if thread isn't overflowing the number of patterns in the group
            if ( pattNum < nPattInGrp) {

                if (matchingColl != -1) {

                    // Get pattern hit data
                    unsigned char pattHit = hitArray[hitArrayGroupIndices[grp] + n*blockDim.x + threadIdx.x];
                    unsigned char hitPos = ((pattHit >> 2) & 31); // Get superstrip position if strip, or row if pixel. Occupies bits 2-6
                    unsigned char dcBits = (pattHit & 3);
                    if (dcBits == 3) { dcBits = 2; }
                    if (isPixel) {
                        unsigned char pattPixCol = hitPos/nMaxRows;
                        unsigned char pattPixRow = hitPos%nMaxRows;
                        hitPos = nMaxRows*pattPixCol + pattPixRow;
                    }
                    if ( ((1 << hitPos) & collectionHit[dcBits*nLayers + lyr]) > 0 ) {
                        atomicAdd(&nPattMatches[row],1);
                    }

                } else if ( lyrHashId == -1 ) {
                    atomicAdd(&nPattMatches[row],1);
                }

                __syncthreads();
                // Output matching pattern ids to array
                if (lyr == 0) {
                    if (nPattMatches[row] >= nRequiredMatches) {
                        int i = atomicAdd(nMatches,1);
                        int pattId = ((hitArrayGroupIndices[grp] - hitArrayGroupIndices[0])/nLayers) + pattNum;
                        matchingPattIds[i] = pattId;
                    }
                }

            } // End if ( pattNum < nPattInGrp)

        } // End nLoops

    } // End if (nHashMatches >= nRequiredMatches)

}

__global__ void matchByBlockMulti(const int *hashId_array, const unsigned char *hitArray, const unsigned int *hitArrayGroupIndices, 
                                  const int *hashId, const unsigned int *hashIdEventIndices, const unsigned int *nHits,
                                  const unsigned int *nHitsEventIndices, const unsigned char *hitData, const unsigned int *hitDataEventIndices, 
                                  int *matchingPattIds, int *nMatches, const int eventId, const int *blockBegin, const int *nGroupsInBlock,
                                  const int *groups) {

    const int nLayers = 8;
    const int maxGroupsInBlock = 60;
    const int nRequiredMatches = 7;
    const int nMaxRows = 11;

    __shared__ unsigned int nHashMatches[maxGroupsInBlock]; // Number of group hashIds that match event collection hashIds for each group
    __shared__ int matchingCollections[nLayers]; // Records which collection matches the group hashId of a certain layer

    // Initialise match counters
    if (threadIdx.x < nGroupsInBlock[blockIdx.x]) {
        nHashMatches[threadIdx.x] = 0;
    }
    __syncthreads();

    // Check if any hashIds are wildcards
    int lyr = threadIdx.x%nLayers;
    int nLoops = (nGroupsInBlock[blockIdx.x]*nLayers/blockDim.x) + 1;
    for (int n = 0; n < nLoops; n++) {
        int grpInBlock = (n*blockDim.x + threadIdx.x)/nLayers;
        if (grpInBlock < nGroupsInBlock[blockIdx.x]) {
            // Initialise matchingCollections to -1
            atomicExch(&matchingCollections[n*blockDim.x + threadIdx.x],-1);
            int grp = groups[blockBegin[blockIdx.x] + grpInBlock];
            int lyrHashId = hashId_array[grp*nLayers + lyr];
            // Automatically match if layer is wildcard
            if (lyrHashId == -1) {
                atomicAdd(&nHashMatches[grpInBlock],1);
             }
        }

    }

    __syncthreads();

    // Get each thread to compare one hashId from each group with one collection from
    // an event to check for a match
    int nColl = hashIdEventIndices[eventId+1] - hashIdEventIndices[eventId];
    nLoops = (nGroupsInBlock[blockIdx.x]*nLayers*nColl/blockDim.x) + 1;
    for (int n = 0; n < nLoops; n++) {
        int grpInBlock = (n*blockDim.x + threadIdx.x)/(nColl*nLayers);
        int coll = ((n*blockDim.x + threadIdx.x)/nLayers)%nColl;
        if (grpInBlock < nGroupsInBlock[blockIdx.x]) {
            int grp = groups[blockBegin[blockIdx.x] + grpInBlock];
            int lyrHashId = hashId_array[grp*nLayers + lyr];
            if (lyrHashId != -1) {
                if (hashId[hashIdEventIndices[eventId] + coll] == lyrHashId) {
                    atomicExch(&matchingCollections[grpInBlock*nLayers + lyr],coll);
                    atomicAdd(&nHashMatches[grpInBlock],1);
                }
            }
        }
    }
    __syncthreads();

    int row = threadIdx.x/nLayers;
    for (int i = 0; i < nGroupsInBlock[blockIdx.x]; i++) {
        int grp = groups[blockBegin[blockIdx.x] + i];
        int lyrHashId = hashId_array[grp*nLayers + lyr];
        if (nHashMatches[i] >= nRequiredMatches) {

            // Loop through collection hits to find collection hit data
            int matchingColl = matchingCollections[i*nLayers + lyr];
            const unsigned char *pHitData = &hitData[hitDataEventIndices[eventId]];
            for (int coll = 0; coll < matchingColl; coll++) {
                pHitData += nHits[nHitsEventIndices[eventId] + coll];
            }

            // Loop as many times as necessary for all threads to cover all patterns
            int nPattInGrp = (hitArrayGroupIndices[grp + 1] - hitArrayGroupIndices[grp])/nLayers;
            nLoops = ((nPattInGrp*nLayers)/blockDim.x) + 1;
            extern __shared__ unsigned int nPattMatches[];

            for (int n = 0; n < nLoops; n++) {

                int pattNum = n*blockDim.x/nLayers + row;

                // Initialise nPattMatches to zero
                if (lyr == 0) {
                    nPattMatches[row] = 0;
                }
                __syncthreads();

                // Only continue if thread isn't overflowing the number of patterns in the group
                if ( pattNum < nPattInGrp) {

                    if (matchingColl != -1) {
    
                        // Get pattern hit data
                        unsigned char pattHit = hitArray[hitArrayGroupIndices[grp] + n*blockDim.x + threadIdx.x];
                        // Decode pattern hit data
                        unsigned char dontCareBitmask = pattHit & 3;
                        if (dontCareBitmask == 2) { dontCareBitmask = 3; } 
                        unsigned char pattHitPos = ((pattHit >> 2) & 31);

                        // Loop through hits
                        for (int hit = 0; hit < nHits[nHitsEventIndices[eventId] + matchingColl]; hit++) {
                            unsigned char eventHitPos = (*(pHitData + hit) & 127);
                            unsigned char eventIsPixel = ((*(pHitData + hit) >> 7) & 1);
                            // Check if pixel or strip
                            if (eventIsPixel) {
                                // Pixel - decode pixel column number
                                unsigned char eventPixCol = (eventHitPos & 3);
                                unsigned char pattPixCol = pattHitPos/nMaxRows;
                                if ( eventPixCol == pattPixCol ) {
                                    // If pixel columns match, decode pixel row, mask with don't care bits and check
                                    // for a match
                                    unsigned char eventPixRow = (((eventHitPos >> 2) & 31) | dontCareBitmask);
                                    unsigned char pattPixRow = (pattHitPos%nMaxRows | dontCareBitmask);
                                    if ( eventPixRow == pattPixRow ) {
                                        atomicAdd(&nPattMatches[row],1);
                                        break;
                                    }
                                }
                            } else {
                                // Strip - decode superstrip values, mask with pattern don't care bits and check
                                // for a match
                                unsigned char eventSuperstrip = (((eventHitPos >> 2) & 31) | dontCareBitmask);
                                unsigned char pattSuperstrip = (pattHitPos | dontCareBitmask);
                                if ( eventSuperstrip == pattSuperstrip ) {
                                    atomicAdd(&nPattMatches[row],1);
                                    break;
                                }
                            }
                        } // End loop over hits

                    } else if ( lyrHashId == -1) {
                        atomicAdd(&nPattMatches[row],1);
                    }

                    __syncthreads();
                    // Output matching pattern ids to array
                    if (lyr == 0) {
                        if (nPattMatches[row] >= nRequiredMatches) {
                            int i = atomicAdd(nMatches,1);
                            int pattId = ((hitArrayGroupIndices[grp] - hitArrayGroupIndices[0])/nLayers) + pattNum;
                            matchingPattIds[i] = pattId;
                        }
                    }

                } // End if ( pattNum < nPattInGrp)

            } // End nLoops

        } // End if (nHashMatches >= nRequiredMatches)
    } // End loop over groups

}

__global__ void matchByLayer(const int *hashId_array, const unsigned char *hitArray, const unsigned int *hitArrayGroupIndices, 
                             const int *hashId, const unsigned int *hashIdEventIndices, const unsigned int *nHits,
                             const unsigned int *nHitsEventIndices, const unsigned char *hitData, const unsigned int *hitDataEventIndices, 
                             int *matchingPattIds, int *nMatches, const int eventId) {
    const int nLayers = 8;
    const int nRequiredMatches = 7;
    const int nMaxRows = 11;
    int grp = blockIdx.x;

    __shared__ unsigned int nHashMatches;
    __shared__ unsigned int nWildcards;

    if (threadIdx.x == 0) {
        nHashMatches = 0;
        nWildcards = 0;
    }
    __syncthreads();

    int lyrHashId = hashId_array[grp*nLayers + threadIdx.x%nLayers];
    // Get first nLayers threads to check if any hashIds are wildcards
    if (threadIdx.x < nLayers) {
        if (lyrHashId == -1) {
            // Automatically match if layer is wildcard
            atomicAdd(&nHashMatches,1);
            atomicAdd(&nWildcards,1);
         }
    }

    // Get each thread to compare one hashId with one collection from
    // an event to check for a match
    int nColl = hashIdEventIndices[eventId+1] - hashIdEventIndices[eventId];
    int nLoops = ((nLayers*nColl)/blockDim.x) + 1;
    for (int n = 0; n < nLoops; n++) {
        int coll = (n*blockDim.x + threadIdx.x)/nLayers;
        if (coll < nColl) {
            if (lyrHashId != -1) {
                if (hashId[hashIdEventIndices[eventId] + coll] == lyrHashId) {
                    atomicAdd(&nHashMatches,1);
                }
            }
        }
    }
    __syncthreads();

    // If there are enough hashId matches, loop through patterns in group
    if (nHashMatches >= nRequiredMatches) {
        int nPattInGrp = (hitArrayGroupIndices[grp + 1] - hitArrayGroupIndices[grp])/nLayers;
        extern __shared__ unsigned int nPattMatches[];

        // Initialise nPattMatches to zero
        nLoops = nPattInGrp/blockDim.x + 1;
        for (int n= 0; n < nLoops; n++) {
            int pattNum = n*blockDim.x + threadIdx.x;
            if (pattNum < nPattInGrp) {
                nPattMatches[pattNum] = 0;
            }
        }
        __syncthreads();

        // Loop as many times as necessary for all threads to cover all patterns
        nLoops = ((nPattInGrp*nLayers)/blockDim.x) + 1;
        for (int n = 0; n < nLoops; n++) {
            int pattNum = (n*blockDim.x + threadIdx.x)%nPattInGrp;
            int lyr = (n*blockDim.x + threadIdx.x)/nPattInGrp;

            // Only continue if thread isn't overflowing the number of layers
            if ( lyr < nLayers) {
                lyrHashId = hashId_array[grp*nLayers + lyr];

                // Automatically match if wildcard layer
                if (lyrHashId == -1) {
                    atomicAdd(&nPattMatches[pattNum],1);
                } else {
                    // Get pattern hit data
                    unsigned char pattHit = hitArray[hitArrayGroupIndices[grp] + pattNum*nLayers + lyr];// n*blockDim.x + threadIdx.x];
                    // Decode pattern hit data
                    unsigned char dontCareBitmask = pattHit & 3;
                        if (dontCareBitmask == 2) { dontCareBitmask = 3; } 
                    unsigned char pattHitPos = ((pattHit >> 2) & 31);

                    // Loop through collections looking for hashId match
                    int nColl = hashIdEventIndices[eventId+1] - hashIdEventIndices[eventId];
                    const unsigned char *pHitData = &hitData[hitDataEventIndices[eventId]];
                    for (int coll = 0; coll < nColl; coll++) {
                        if (hashId[hashIdEventIndices[eventId] + coll] == lyrHashId) {
                            // Once the matching collection has been found, loop through hits
                            for (int hit = 0; hit < nHits[nHitsEventIndices[eventId] + coll]; hit++) {
                                unsigned char eventHitPos = (*(pHitData + hit) & 127);
                                unsigned char eventIsPixel = ((*(pHitData + hit) >> 7) & 1);
                                // Check if pixel or strip
                                if (eventIsPixel) {
                                    // Pixel - decode pixel column number
                                    unsigned char eventPixCol = (eventHitPos & 3);
                                    unsigned char pattPixCol = pattHitPos/nMaxRows;
                                    if ( eventPixCol == pattPixCol ) {
                                        // If pixel columns match, decode pixel row, mask with don't care bits and check
                                        // for a match
                                        unsigned char eventPixRow = (((eventHitPos >> 2) & 31) | dontCareBitmask);
                                        unsigned char pattPixRow = (pattHitPos%nMaxRows | dontCareBitmask);
                                        if ( eventPixRow == pattPixRow ) {
                                                atomicAdd(&nPattMatches[pattNum],1);
                                                break;
                                        }
                                    }
                                } else {
                                    // Strip - decode superstrip values, mask with pattern don't care bits and check
                                    // for a match
                                    unsigned char eventSuperstrip = (((eventHitPos >> 2) & 31) | dontCareBitmask);
                                    unsigned char pattSuperstrip = (pattHitPos | dontCareBitmask);
                                    if ( eventSuperstrip == pattSuperstrip ) {
                                        atomicAdd(&nPattMatches[pattNum],1);
                                        break;
                                    }
                                }
                            }
                            break; // Break once the matching collection has been checked
                        }
                        pHitData += nHits[nHitsEventIndices[eventId] + coll];
                    }
                }
            } // End if lyr < nLayers
        } // End loop over patterns

        __syncthreads();
        // Output matching pattern ids to array
        nLoops = nPattInGrp/blockDim.x + 1;
        for (int n = 0; n < nLoops; n++) {
            int pattNum = n*blockDim.x + threadIdx.x;
            if (pattNum < nPattInGrp) {
                if (nPattMatches[pattNum] >= nRequiredMatches) {
                    int i = atomicAdd(nMatches,1);
                    int pattId = ((hitArrayGroupIndices[grp] - hitArrayGroupIndices[0])/nLayers) + pattNum;
                    matchingPattIds[i] = pattId;
                }
            }
        }

    }
}

