#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "eventReader.h"
#include "patternReader.h"
#include "matchResults.h"
#include "tools.h"

#include "gpuMatcher.h"

using namespace std;

void createGpuContext(const PatternContainer& p, const EventContainer& e, GpuContext& ctx) {
    hipError_t err = hipSuccess;

    // For all group/event begins, calculate indices that are pointed to
    vector<unsigned int> h_hitArrayGroupIndices = pointerToIndex(p.hitArrayGroupBegin, p.hitArray);
    vector<unsigned int> h_hashIdEventIndices = pointerToIndex(e.hashIdEventBegin, e.hashId);
    vector<unsigned int> h_nHitsEventIndices = pointerToIndex(e.nHitsEventBegin, e.nHits);
    vector<unsigned int> h_hitDataEventIndices = pointerToIndex(e.hitDataEventBegin, e.hitData);

    // Calculate size for all arrays that will be transferred
    size_t hashId_array_size = sizeof(int)*p.hashId_array.size();
    size_t hitArray_size = sizeof(unsigned char)*p.hitArray.size();
    size_t hitArrayGroupIndices_size = sizeof(unsigned int)*h_hitArrayGroupIndices.size();
    size_t hashId_size = sizeof(int)*e.hashId.size();
    size_t hashIdEventIndices_size = sizeof(unsigned int)*h_hashIdEventIndices.size();
    size_t nHits_size = sizeof(unsigned int)*e.nHits.size();
    size_t nHitsEventIndices_size = sizeof(unsigned int)*h_nHitsEventIndices.size();
    size_t hitData_size = sizeof(unsigned char)*e.hitData.size();
    size_t hitDataEventIndices_size = sizeof(unsigned int)*h_hitDataEventIndices.size();
    size_t matchingPattIds_size = sizeof(int)*10000;

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Allocate space for arrays on device
    err = hipMalloc((void ** )&ctx.d_hashId_array, hashId_array_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hashId_array\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitArray, hitArray_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitArray\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitArrayGroupIndices, hitArrayGroupIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitArrayGroupIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hashId, hashId_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hashId\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hashIdEventIndices, hashIdEventIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hashIdEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_nHits, nHits_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_nHits\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_nHitsEventIndices, nHitsEventIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_nHitsEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitData, hitData_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitData\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_hitDataEventIndices, hitDataEventIndices_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_hitDataEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_matchingPattIds, matchingPattIds_size);
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_matchingPattIds\n" << hipGetErrorString(err) << endl;
    err = hipMalloc((void ** )&ctx.d_nMatches, sizeof(int));
    if (err != hipSuccess) cerr << "Error: device memory not successfully allocated for d_nMatches\n" << hipGetErrorString(err) << endl;

    // Copy input arrays to device
    err = hipMemcpy(ctx.d_hashId_array, &p.hashId_array[0], hashId_array_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hashId_array not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitArray, &p.hitArray[0], hitArray_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitArray not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitArrayGroupIndices, &h_hitArrayGroupIndices[0], hitArrayGroupIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitArrayGroupIndices not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hashId, &e.hashId[0], hashId_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hashId not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hashIdEventIndices, &h_hashIdEventIndices[0], hashIdEventIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hashIdEventIndices not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_nHits, &e.nHits[0], nHits_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: nHits not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_nHitsEventIndices, &h_nHitsEventIndices[0], nHitsEventIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: nHitsEventIndices not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitData, &e.hitData[0], hitData_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitData not copied to device\n" << hipGetErrorString(err) << endl;
    err = hipMemcpy(ctx.d_hitDataEventIndices, &h_hitDataEventIndices[0], hitDataEventIndices_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) cerr << "Error: hitDataEventIndices not copied to device\n" << hipGetErrorString(err) << endl;

    // Initialise output arrays
    err = hipMemset(ctx.d_matchingPattIds, 0, matchingPattIds_size);
    if (err != hipSuccess) cerr << "Error: d_matchingPattIds not initialised to zero" << endl;
    err = hipMemset(ctx.d_nMatches, 0, sizeof(int));
    if (err != hipSuccess) cerr << "Error: d_matchingPattIds not initialised to zero" << endl;

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;
    cout << "Allocated and copied arrays to device in " << msecTotal << " ms" << endl;
};


void runMatchByBlock(const PatternContainer& p, const EventContainer& e, GpuContext& ctx, MatchResults& mr, int threadsPerBlock) {
    hipError_t err = hipSuccess;

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Calculate number of blocks required
    int blocksPerGrid = p.header.nGroups;

    // Run kernel for each event
    int nPattMatchesSize = threadsPerBlock/p.header.nLayers*sizeof(unsigned int);
    for (int i = 0; i < e.header.nEvents; i++ ) {
        matchByBlock<<<blocksPerGrid, threadsPerBlock, nPattMatchesSize>>>(ctx.d_hashId_array, ctx.d_hitArray, ctx.d_hitArrayGroupIndices,
                                                                           ctx.d_hashId, ctx.d_hashIdEventIndices, ctx.d_nHits,
                                                                           ctx.d_nHitsEventIndices, ctx.d_hitData, ctx.d_hitDataEventIndices,
                                                                           ctx.d_matchingPattIds, ctx.d_nMatches, p.header.nGroups, p.header.nLayers, i);
    }
    hipDeviceSynchronize();

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;
    cout << "Ran kernel " << e.header.nEvents << " times in " << msecTotal << " ms" << endl;
    float msecPerEvent = msecTotal/e.header.nEvents;
    cout << "Average matchByBlock kernel time with " << threadsPerBlock << " threads is " << msecPerEvent << " ms" << endl;

    // Copy result back to host memory
    err = hipMemcpy(&mr.nMatches, ctx.d_nMatches, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) cerr << "Error: d_nMatches not copied from device to host" << endl;
    mr.patternIds.resize(mr.nMatches);
    err = hipMemcpy(&mr.patternIds[0], ctx.d_matchingPattIds, mr.nMatches*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) cerr << "Error: d_matchingPattIds not copied from device to host" << endl;

};

void runMatchByLayer(const PatternContainer& p, const EventContainer& e, GpuContext& ctx, MatchResults& mr, int threadsPerBlock) {
    hipError_t err = hipSuccess;

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Calculate number of blocks required
    int blocksPerGrid = p.header.nGroups;

    // Run kernel for each event
    int nPattMatchesSize = p.nPattInGrp[0]*sizeof(unsigned int);
    for (int i = 0; i < e.header.nEvents; i++ ) {
        matchByLayer<<<blocksPerGrid, threadsPerBlock, nPattMatchesSize>>>(ctx.d_hashId_array, ctx.d_hitArray, ctx.d_hitArrayGroupIndices,
                                                                           ctx.d_hashId, ctx.d_hashIdEventIndices, ctx.d_nHits,
                                                                           ctx.d_nHitsEventIndices, ctx.d_hitData, ctx.d_hitDataEventIndices,
                                                                           ctx.d_matchingPattIds, ctx.d_nMatches, p.header.nGroups, p.header.nLayers, i);
    }
    hipDeviceSynchronize();

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;
    cout << "Ran kernel " << e.header.nEvents << " times in " << msecTotal << " ms" << endl;
    float msecPerEvent = msecTotal/e.header.nEvents;
    cout << "Average matchByLayer kernel time with " << threadsPerBlock << " threads is " << msecPerEvent << " ms" << endl;

    // Copy result back to host memory
    err = hipMemcpy(&mr.nMatches, ctx.d_nMatches, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) cerr << "Error: d_nMatches not copied from device to host" << endl;
    mr.patternIds.resize(mr.nMatches);
    err = hipMemcpy(&mr.patternIds[0], ctx.d_matchingPattIds, mr.nMatches*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) cerr << "Error: d_matchingPattIds not copied from device to host" << endl;

};


void deleteGpuContext(GpuContext& ctx) {
    hipError_t err = hipSuccess;

    // Create timer events
    hipEvent_t start;
    err = hipEventCreate(&start);
    if (err != hipSuccess) cerr << "Error: failed to create timer start event\n" << hipGetErrorString(err) << endl;
    hipEvent_t stop;
    err = hipEventCreate(&stop);
    if (err != hipSuccess) cerr << "Error: failed to create timer stop event\n" << hipGetErrorString(err) << endl;

    // Record timer start event
    err = hipEventRecord(start, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer start event\n" << hipGetErrorString(err) << endl;

    // Free device memory
    err = hipFree(ctx.d_hashId_array);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hashId_array\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitArray);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitArray\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitArrayGroupIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitArrayGroupIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hashId);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hashId\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hashIdEventIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hashIdEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_nHits);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_nHits\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_nHitsEventIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_nHitsEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitData);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitData\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_hitDataEventIndices);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_hitDataEventIndices\n" << hipGetErrorString(err) << endl;
    err = hipFree(ctx.d_matchingPattIds);
    if (err != hipSuccess) cerr << "Error: failed to free device memory for d_matchingPattIds\n" << hipGetErrorString(err) << endl;

    // Record timer stop event
    err = hipEventRecord(stop, NULL);
    if (err != hipSuccess) cerr << "Error: failed to record timer stop event\n" << hipGetErrorString(err) << endl;
    err = hipEventSynchronize(stop);
    if (err != hipSuccess) cerr << "Error: failed to synchronize on stop event\n" << hipGetErrorString(err) << endl;

    // Calculate elapsed time
    float msecTotal = 0.0f;
    err = hipEventElapsedTime(&msecTotal, start, stop);
    if (err != hipSuccess) cerr << "Error: failed to get elapsed time between events\n" << hipGetErrorString(err) << endl;

    cout << "Freed device memory in " << msecTotal << " ms" << endl;

};

__global__ void matchByBlock(const int *hashId_array, const unsigned char *hitArray, const unsigned int *hitArrayGroupIndices, 
                             const int *hashId, const unsigned int *hashIdEventIndices, const unsigned int *nHits,
                             const unsigned int *nHitsEventIndices, const unsigned char *hitData, const unsigned int *hitDataEventIndices, 
                             int *matchingPattIds, int *nMatches, const int nGroups, const int nLayers, const int eventId) {
    int nRequiredMatches = 7;
    int nMaxRows = 22;
    int grp = blockIdx.x;
    int lyr = threadIdx.x%nLayers;
    int row = threadIdx.x/nLayers;

    __shared__ unsigned int nHashMatches;
    __shared__ unsigned int nWildcards;

    int lyrHashId = hashId_array[grp*nLayers + lyr];
    // Get first nLayers threads to check the group hashIds and check if they are
    // a potential match for this event
    if (threadIdx.x < nLayers) {
        if (lyrHashId == -1) {
            // Automatically match if layer is wildcard
            atomicAdd(&nHashMatches,1);
            atomicAdd(&nWildcards,1);
        } else {
            // Otherwise loop through collections looking for match
            int nColl = hashIdEventIndices[eventId+1] - hashIdEventIndices[eventId];
            for (int coll = 0; coll < nColl; coll++) {
                if (hashId[hashIdEventIndices[eventId] + coll] == lyrHashId) {
                    atomicAdd(&nHashMatches,1);
                    // Break out of collection if a match is found
                    break;
                }
            }
        }

    }
    __syncthreads();

    // If there are enough hashId matches, loop through patterns in group
    if (nHashMatches >= nRequiredMatches) {
        int nPattInGrp = (hitArrayGroupIndices[grp + 1] - hitArrayGroupIndices[grp])/nLayers;
        int nLoops = ((nPattInGrp*nLayers)/blockDim.x) + 1;
        // Loop as many times as necessary for all threads to cover all patterns
        extern __shared__ unsigned int nPattMatches[];

        for (int n = 0; n < nLoops; n++) {
            int pattNum = n*blockDim.x/nLayers + row;

            // Only continue if thread isn't overflowing the number of patterns in the group
            if ( pattNum < nPattInGrp) {
                // Initialise nPattMatches to zero
                if (lyr == 0) {
                    nPattMatches[row] = 0;
                }
                __syncthreads();
    
                // Automatically match if wildcard layer
                if (lyrHashId == -1) {
                    atomicAdd(&nPattMatches[row],1);
                } else {
                    // Get pattern hit data
                    unsigned char pattHit = hitArray[hitArrayGroupIndices[grp] + n*blockDim.x + threadIdx.x];
                    // Decode pattern hit data
                    unsigned char dontCareBitmask = pattHit & 3;
                    unsigned char pattHitPos = ((pattHit >> 2) & 31);

                    // Loop through collections looking for hashId match
                    int nColl = hashIdEventIndices[eventId+1] - hashIdEventIndices[eventId];
                    const unsigned char *pHitData = &hitData[hitDataEventIndices[eventId]];
                    for (int coll = 0; coll < nColl; coll++) {
                        if (hashId[hashIdEventIndices[eventId] + coll] == lyrHashId) {
                            // Once the matching collection has been found, loop through hits
                            for (int hit = 0; hit < nHits[nHitsEventIndices[eventId] + coll]; hit++) {
                                unsigned char eventHitPos = (*(pHitData + hit) & 127);
                                unsigned char eventIsPixel = ((*(pHitData + hit) >> 7) & 1);
                                // Check if pixel or strip
                                if (eventIsPixel) {
                                    // Pixel - decode pixel column number
                                    unsigned char eventPixCol = (eventHitPos & 3);
                                    unsigned char pattPixCol = pattHitPos/nMaxRows;
                                    if ( eventPixCol == pattPixCol ) {
                                        // If pixel columns match, decode pixel row, mask with don't care bits and check
                                        // for a match
                                        unsigned char eventPixRow = (((eventHitPos >> 2) & 31) | dontCareBitmask);
                                        unsigned char pattPixRow = (pattHitPos%nMaxRows | dontCareBitmask);
                                        if ( eventPixRow == pattPixRow ) {
                                                atomicAdd(&nPattMatches[row],1);
                                                break;
                                        }
                                    }
                                } else {
                                    // Strip - decode superstrip values, mask with pattern don't care bits and check
                                    // for a match
                                    unsigned char eventSuperstrip = (((eventHitPos >> 2) & 31) | dontCareBitmask);
                                    unsigned char pattSuperstrip = (pattHitPos | dontCareBitmask);
                                    if ( eventSuperstrip == pattSuperstrip ) {
                                        atomicAdd(&nPattMatches[row],1);
                                        break;
                                    }
                                }
                            }
                            break; // Break once the matching collection has been checked
                        }
                        pHitData += nHits[nHitsEventIndices[eventId] + coll];
                    }
                }
                __syncthreads();
                // Output matching pattern ids to array
                if (lyr == 0) {
                    if (nPattMatches[row] >= nRequiredMatches) {
                        int i = atomicAdd(nMatches,1);
                        int pattId = ((hitArrayGroupIndices[grp] - hitArrayGroupIndices[0])/nLayers) + pattNum;
                        matchingPattIds[i] = pattId;
                    }
                }
            }
        } // End loop over patterns

    }
}

__global__ void matchByLayer(const int *hashId_array, const unsigned char *hitArray, const unsigned int *hitArrayGroupIndices, 
                             const int *hashId, const unsigned int *hashIdEventIndices, const unsigned int *nHits,
                             const unsigned int *nHitsEventIndices, const unsigned char *hitData, const unsigned int *hitDataEventIndices, 
                             int *matchingPattIds, int *nMatches, const int nGroups, const int nLayers, const int eventId) {
    int nRequiredMatches = 7;
    int nMaxRows = 22;
    int grp = blockIdx.x;

    __shared__ unsigned int nHashMatches;
    __shared__ unsigned int nWildcards;

    // Get first nLayers threads to check the group hashIds and check if they are
    // a potential match for this event
    if (threadIdx.x < nLayers) {
        int grpCheckHashId = hashId_array[grp*nLayers + threadIdx.x];
        if (grpCheckHashId == -1) {
            // Automatically match if layer is wildcard
            atomicAdd(&nHashMatches,1);
            atomicAdd(&nWildcards,1);
        } else {
            // Otherwise loop through collections looking for match
            int nColl = hashIdEventIndices[eventId+1] - hashIdEventIndices[eventId];
            for (int coll = 0; coll < nColl; coll++) {
                if (hashId[hashIdEventIndices[eventId] + coll] == grpCheckHashId) {
                    atomicAdd(&nHashMatches,1);
                    // Break out of collection if a match is found
                    break;
                }
            }
        }

    }
    __syncthreads();

    // If there are enough hashId matches, loop through patterns in group
    if (nHashMatches >= nRequiredMatches) {
        int nPattInGrp = (hitArrayGroupIndices[grp + 1] - hitArrayGroupIndices[grp])/nLayers;
        extern __shared__ unsigned int nPattMatches[];

        // Initialise nPattMatches to zero
        int mLoops = nPattInGrp/blockDim.x + 1;
        for (int m = 0; m < mLoops; m++) {
            int pattNum = m*blockDim.x + threadIdx.x;
            if (pattNum < nPattInGrp) {
                nPattMatches[pattNum] = 0;
            }
        }
        __syncthreads();

        // Loop as many times as necessary for all threads to cover all patterns
        int nLoops = ((nPattInGrp*nLayers)/blockDim.x) + 1;
        for (int n = 0; n < nLoops; n++) {
            int pattNum = (n*blockDim.x + threadIdx.x)%nPattInGrp;
            int lyr = (n*blockDim.x + threadIdx.x)/nPattInGrp;

            // Only continue if thread isn't overflowing the number of layers
            if ( lyr < nLayers) {
                int lyrHashId = hashId_array[grp*nLayers + lyr];

                // Automatically match if wildcard layer
                if (lyrHashId == -1) {
                    atomicAdd(&nPattMatches[pattNum],1);
                } else {
                    // Get pattern hit data
                    unsigned char pattHit = hitArray[hitArrayGroupIndices[grp] + pattNum*nLayers + lyr];// n*blockDim.x + threadIdx.x];
                    // Decode pattern hit data
                    unsigned char dontCareBitmask = pattHit & 3;
                    unsigned char pattHitPos = ((pattHit >> 2) & 31);

                    // Loop through collections looking for hashId match
                    int nColl = hashIdEventIndices[eventId+1] - hashIdEventIndices[eventId];
                    const unsigned char *pHitData = &hitData[hitDataEventIndices[eventId]];
                    for (int coll = 0; coll < nColl; coll++) {
                        if (hashId[hashIdEventIndices[eventId] + coll] == lyrHashId) {
                            // Once the matching collection has been found, loop through hits
                            for (int hit = 0; hit < nHits[nHitsEventIndices[eventId] + coll]; hit++) {
                                unsigned char eventHitPos = (*(pHitData + hit) & 127);
                                unsigned char eventIsPixel = ((*(pHitData + hit) >> 7) & 1);
                                // Check if pixel or strip
                                if (eventIsPixel) {
                                    // Pixel - decode pixel column number
                                    unsigned char eventPixCol = (eventHitPos & 3);
                                    unsigned char pattPixCol = pattHitPos/nMaxRows;
                                    if ( eventPixCol == pattPixCol ) {
                                        // If pixel columns match, decode pixel row, mask with don't care bits and check
                                        // for a match
                                        unsigned char eventPixRow = (((eventHitPos >> 2) & 31) | dontCareBitmask);
                                        unsigned char pattPixRow = (pattHitPos%nMaxRows | dontCareBitmask);
                                        if ( eventPixRow == pattPixRow ) {
                                                atomicAdd(&nPattMatches[pattNum],1);
                                                break;
                                        }
                                    }
                                } else {
                                    // Strip - decode superstrip values, mask with pattern don't care bits and check
                                    // for a match
                                    unsigned char eventSuperstrip = (((eventHitPos >> 2) & 31) | dontCareBitmask);
                                    unsigned char pattSuperstrip = (pattHitPos | dontCareBitmask);
                                    if ( eventSuperstrip == pattSuperstrip ) {
                                        atomicAdd(&nPattMatches[pattNum],1);
                                        break;
                                    }
                                }
                            }
                            break; // Break once the matching collection has been checked
                        }
                        pHitData += nHits[nHitsEventIndices[eventId] + coll];
                    }
                }
            } // End if lyr < nLayers
        } // End loop over patterns

        __syncthreads();
        //int mLoops = nPattInGrp/blockDim.x + 1;
        // Output matching pattern ids to array
        for (int m = 0; m < mLoops; m++) {
            int pattNum = m*blockDim.x + threadIdx.x;
            if (pattNum < nPattInGrp) {
                if (nPattMatches[pattNum] >= nRequiredMatches) {
                    int i = atomicAdd(nMatches,1);
                    int pattId = ((hitArrayGroupIndices[grp] - hitArrayGroupIndices[0])/nLayers) + pattNum;
                    matchingPattIds[i] = pattId;
                }
            }
        }

    }
}

